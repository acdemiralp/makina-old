#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2 , launch_index, rtLaunchIndex, );
rtDeclareVariable(float3, color       ,              , );

rtBuffer<float4, 2> output;

RT_PROGRAM void draw_color()
{
  output[launch_index] = make_float4(color, 0.f);
}